#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <stdio.h>
#include "simulator.h"
#include <stdlib.h>

#define softeningSquared 0.01f		// original plumer softener is 0.025. here the value is square of it.
#define damping 1.0f				// 0.999f
#define G 1
//6.67418478E-11


//EXTERN_DLL_EXPORT
//__host__ __device__ Particle updateSingleParticle(int particleInd, Particle* pdata, float step, int nbodies) {
//	Particle p = pdata[particleInd];
//
//	// update gravity (accumulation): naive big loop
//	float3 acc = { 0.0f, 0.0f, 0.0f };
//	float distSqr, distCube, s;
//
//	Particle r;
//
//	for (int i = 0; i < nbodies; i++)
//	{
//		r = pdata[i];
//
//		r.pos.x -= p.pos.x;
//		r.pos.y -= p.pos.y;
//		r.pos.z -= p.pos.z;
//
//		distSqr = r.pos.x * r.pos.x + r.pos.y * r.pos.y + r.pos.z * r.pos.z;
//		distSqr += softeningSquared;
//
//		float dist = sqrtf(distSqr);
//		distCube = dist * dist * dist + 0.001;
//
//		s = r.weight / distCube;
//
//		acc.x += r.pos.x * s;
//		acc.y += r.pos.y * s;
//		acc.z += r.pos.z * s;
//	}
//
//	// update velocity with above acc
//	p.vel.x += acc.x * step;
//	p.vel.y += acc.y * step;
//	p.vel.z += acc.z * step;
//
//	p.vel.x *= damping;
//	p.vel.y *= damping;
//	p.vel.z *= damping;
//
//	// update position
//	p.pos.x += p.vel.x * step;
//	p.pos.y += p.vel.y * step;
//	p.pos.z += p.vel.z * step;
//
//	return p;
//}

EXTERN_DLL_EXPORT
__host__ __device__ Particle updateSingleParticle(int particleInd, Particle* pdata, float step, int nbodies) {
	Particle p = pdata[particleInd];

	// update gravity (accumulation): naive big loop
	float3 acc = { 0.0f, 0.0f, 0.0f };
	float distSqr, distCube, s;

	Particle r;
	double dx, dy, dz;

	for (int i = 0; i < nbodies; i++)
	{
		if (i == particleInd)
			continue;
		r = pdata[i];

		dx = p.pos.x - r.pos.x;
		dy = p.pos.y - r.pos.y;
		dz = p.pos.z - r.pos.z;

		distSqr = (dx * dx + dy * dy + dz * dz + softeningSquared);

		float dist = sqrtf(distSqr);
		float magi = (G * r.weight) / (dist * dist * dist);
		acc.x -= magi * dx;
		acc.y -= magi * dy;
		acc.z -= magi * dz;
	}

	// update velocity with above acc
	p.vel.x += acc.x * step;
	p.vel.y += acc.y * step;
	p.vel.z += acc.z * step;

	/*p.vel.x *= damping;
	p.vel.y *= damping;
	p.vel.z *= damping;*/

	// update position
	p.pos.x += p.vel.x * step;
	p.pos.y += p.vel.y * step;
	p.pos.z += p.vel.z * step;

	return p;
}


__global__ void galaxyKernel(Particle* pdata, float step, int nbodies)
{
	// index for vertex (pos)
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x >= nbodies) {
		return;
	}
	
	Particle p = updateSingleParticle(x, pdata, step, nbodies);
	
	// thread synch
	__syncthreads();

	// update global memory with update value (position, velocity)
	pdata[x] = p;
}

//__global__ void galaxyKernel(float4* pdata, float step, int nbodies)
//{
//	// index for vertex (pos)
//	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
//	unsigned int y = blockDim.x * gridDim.x + x;
//
//	if (x >= nbodies || y >= nbodies) {
//		return;
//	}
//	auto el = pdata[1];
//
//	float4 p = pdata[x];
//	float4 v = pdata[y];
//
//	// update gravity (accumulation): naive big loop
//	float3 acc = { 0.0f, 0.0f, 0.0f };
//	float distSqr, distCube, s;
//
//	unsigned int ni = 0;
//
//	float4 r;
//
//	for (int i = 0; i < nbodies; i++)
//	{
//		r = pdata[i];
//
//		r.x -= p.x;
//		r.y -= p.y;
//		r.z -= p.z;
//
//		distSqr = r.x * r.x + r.y * r.y + r.z * r.z;
//		distSqr += softeningSquared;
//
//		float dist = sqrtf(distSqr);
//		distCube = dist * dist * dist;
//
//		s = r.w / distCube;
//
//		acc.x += r.x * s;
//		acc.y += r.y * s;
//		acc.z += r.z * s;
//	}
//
//	// update velocity with above acc
//	v.x += acc.x * step;
//	v.y += acc.y * step;
//	v.z += acc.z * step;
//
//	// update position
//	p.x += v.x * step;
//	p.y += v.y * step;
//	p.z += v.z * step;
//
//	// thread synch
//	__syncthreads();
//
//	// update global memory with update value (position, velocity)
//	pdata[x] = p;
//	pdata[y] = v;
//}

int updateSimulationCuda(SimulationData* data, float timeStep) {
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}


	Particle* deviceParticleData;

	cudaStatus = hipMalloc((void**)&deviceParticleData, data->nbodies * sizeof(Particle));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	hipMemcpy(deviceParticleData, data->particleData, data->nbodies * sizeof(Particle), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMemcopyHostToDevice failed!");
		goto Error;
	}
	
	galaxyKernel <<<256, 256 >>> (deviceParticleData, timeStep, data->nbodies);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "galaxyKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(data->particleData, deviceParticleData, data->nbodies * sizeof(Particle), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(deviceParticleData);
	return cudaStatus != hipSuccess? 1: 0;
}

int updateSimulationC(SimulationData* data, float timeStep) {
	Particle* particles = (Particle*)malloc(data->nbodies * sizeof(Particle));
	if (!particles)
		return 1;
	for (int i = 0; i < data->nbodies; i++) {
		particles[i] = updateSingleParticle(i, data->particleData, timeStep, data->nbodies);
	}
	free(data->particleData);
	data->particleData = particles;
	return 0;
}